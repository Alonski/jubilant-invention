#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define HISTOGRAM_SIZE 256

__global__ void computeOnGPU(int *histogram, int *numbers, int numElements)
{
    int g_index = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ int shared_numbers[HISTOGRAM_SIZE];
    shared_numbers[threadIdx.x] = 0;

    if (g_index < numElements)
        atomicAdd(&shared_numbers[numbers[g_index]], 1);

    __syncthreads();
    atomicAdd(&histogram[threadIdx.x], shared_numbers[threadIdx.x]);
    __syncthreads();
}

int *calculateHistogramCUDA(int *numbers, int numElements)
{
    int blocks = numElements / HISTOGRAM_SIZE;
    int *local_histogram = 0, *local_numbers = 0;
    int *histogram = (int *)malloc((HISTOGRAM_SIZE + 1) * sizeof(int));

    // Used to save the error returned from CUDA
    hipError_t err = hipSuccess;

    if (numElements % HISTOGRAM_SIZE != 0)
    {
        blocks++;
    }

    // Allocate space for histogram on GPU
    err = hipMalloc((void **)&local_histogram, (HISTOGRAM_SIZE + 1) * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device mem - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Set empty histogram from the host to GPU
    err = hipMemset(local_histogram, 0, (HISTOGRAM_SIZE + 1) * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to set device mem - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate space for numbers on GPU
    err = hipMalloc((void **)&local_numbers, numElements * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device mem - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy numbers array from memory in host to GPU
    err = hipMemcpy(local_numbers, numbers, numElements * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy from host - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Calculate histogram array on GPU
    computeOnGPU<<<blocks, HISTOGRAM_SIZE>>>(local_histogram, local_numbers, numElements);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch GPU add -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy back histogram from GPU to host
    err = hipMemcpy(histogram, local_histogram, (HISTOGRAM_SIZE + 1) * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy from GPU to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    if (hipFree(local_histogram) != hipSuccess)
    {
        fprintf(stderr, "Failed to free histogram memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    if (hipFree(local_numbers) != hipSuccess)
    {
        fprintf(stderr, "Failed to free numbers memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return histogram;
}