#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "proto.h"

__global__ void computeOnGPU(int *histogram, int *data, int numElements)
{
    int g_index = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ int shared_data[HISTOGRAM_SIZE];
    shared_data[threadIdx.x] = 0;

    if (g_index < numElements)
        atomicAdd(&shared_data[data[g_index]], 1);

    __syncthreads();
    atomicAdd(&histogram[threadIdx.x], shared_data[threadIdx.x]);
    __syncthreads();
}

int calculateHistogramCUDA(int *histogram, int *data, int numElements)
{
    // Used to save the error returned from CUDA
    hipError_t err = hipSuccess;

    size_t size = numElements * sizeof(int);

    // Allocate on GPU and copy from the host
    int *d_data, *d_histo;
    err = hipMalloc((void **)&d_data, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device mem - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // copy from the host to GPU
    err = hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy from host - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_histo, HISTOGRAM_SIZE * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device mem - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemset(d_histo, 0, HISTOGRAM_SIZE * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to set device mem - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Initiate Kernel
    int blocks = (numElements + HISTOGRAM_SIZE - 1) / HISTOGRAM_SIZE;
    computeOnGPU<<<blocks, HISTOGRAM_SIZE>>>(d_histo, d_data, numElements);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vector add -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy from GPU to host.
    err = hipMemcpy(histogram, d_histo, HISTOGRAM_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy from GPU to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free memory
    if (hipFree(d_data) != hipSuccess)
    {
        fprintf(stderr, "Failed to free mem - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if (hipFree(d_histo) != hipSuccess)
    {
        fprintf(stderr, "Failed to free data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}
